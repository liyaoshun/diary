#include "hip/hip_runtime.h"
#include "gridSampler.h"
#include "gridSampler.cuh"
#include <stdio.h>
#include <assert.h>

using half = __half;

// alignd with torch Macros.h
// CUDA_MAX_THREADS_PER_BLOCK is same for all architectures currently
constexpr unsigned int CUDA_MAX_THREADS_PER_BLOCK = 1024;
// CUDA_THREADS_PER_BLOCK_FALLBACK is the "canonical fallback" choice of block size.
// 256 is a good number for this fallback and should give good occupancy and
// versatility across all architectures.
constexpr unsigned int CUDA_THREADS_PER_BLOCK_FALLBACK = 256;

#define C10_MAX_THREADS_PER_BLOCK(val) (((val) <= CUDA_MAX_THREADS_PER_BLOCK) ? (val) : CUDA_THREADS_PER_BLOCK_FALLBACK)
#define C10_LAUNCH_BOUNDS_1(max_threads_per_block) __launch_bounds__((C10_MAX_THREADS_PER_BLOCK((max_threads_per_block))))

// aligned with KernelUtils.h
#define CUDA_KERNEL_LOOP(i, n) \
  int64_t _i_n_d_e_x = blockIdx.x * blockDim.x + threadIdx.x;                                \
  for (int i=_i_n_d_e_x; _i_n_d_e_x < (n); _i_n_d_e_x+=blockDim.x * gridDim.x, i=_i_n_d_e_x)


constexpr int CUDA_NUM_THREADS = 1024;
inline int GET_BLOCKS(const int N)
{
  assert(N > 0);
  return (N + CUDA_NUM_THREADS - 1) / CUDA_NUM_THREADS;
}

template <typename scalar_t>
C10_LAUNCH_BOUNDS_1(1024)
__global__ void grid_sampler_2d_kernel(
    int nthreads,
    const scalar_t* inputPtr,
    const scalar_t* gridPtr,      
    scalar_t* const outputPtr,      
    int C,
    int inp_H,
    int inp_W,
    int out_H, // same as grid_H
    int out_W, // same as grid_W
    int inp_sN,
    int inp_sC,
    int inp_sH,
    int inp_sW,
    int grid_sN,
    int grid_sH,
    int grid_sW,
    int grid_sCoor,
    int out_sN,
    int out_sC,
    int out_sH,
    int out_sW,
    torch::detail::GridSamplerInterpolation interpolation_mode,
    torch::detail::GridSamplerPadding padding_mode,
    bool align_corners) {

  CUDA_KERNEL_LOOP(index, nthreads) {
    const int w = index % out_W;
    const int h = (index / out_W) % out_H;
    const int n = index / (out_H * out_W);
    const int grid_offset = n * grid_sN + h * grid_sH + w * grid_sW;

    // get the corresponding input x, y co-ordinates from grid
    scalar_t ix = gridPtr[grid_offset];
    scalar_t iy = gridPtr[grid_offset + grid_sCoor];

    ix = grid_sampler_compute_source_index(ix, inp_W, padding_mode, align_corners);
    iy = grid_sampler_compute_source_index(iy, inp_H, padding_mode, align_corners);

    if (interpolation_mode == torch::detail::GridSamplerInterpolation::Bilinear) {
      // get NE, NW, SE, SW pixel values from (x, y)
      int ix_nw = static_cast<int>(floorf(ix));
      int iy_nw = static_cast<int>(floorf(iy));
      int ix_ne = ix_nw + 1;
      int iy_ne = iy_nw;
      int ix_sw = ix_nw;
      int iy_sw = iy_nw + 1;
      int ix_se = ix_nw + 1;
      int iy_se = iy_nw + 1;

      // get surfaces to each neighbor:
      scalar_t nw = (static_cast<scalar_t>(ix_se) - ix)    * (static_cast<scalar_t>(iy_se) - iy);
      scalar_t ne = (ix    - ix_sw) * (static_cast<scalar_t>(iy_sw) - iy);
      scalar_t sw = (static_cast<scalar_t>(ix_ne) - ix)    * (iy    - iy_ne);
      scalar_t se = (ix    - ix_nw) * (iy    - iy_nw);

      // calculate bilinear weighted pixel value and set output pixel
      auto inp_ptr_NC = inputPtr + n * inp_sN;
      auto out_ptr_NCHW = outputPtr + n * out_sN + h * out_sH + w * out_sW;
      for (int c = 0; c < C; ++c, inp_ptr_NC += inp_sC, out_ptr_NCHW += out_sC) {
        *out_ptr_NCHW = static_cast<scalar_t>(0);
        if (within_bounds_2d(iy_nw, ix_nw, inp_H, inp_W)) {
          *out_ptr_NCHW += inp_ptr_NC[iy_nw * inp_sH + ix_nw * inp_sW] * nw;
        }
        if (within_bounds_2d(iy_ne, ix_ne, inp_H, inp_W)) {
          *out_ptr_NCHW += inp_ptr_NC[iy_ne * inp_sH + ix_ne * inp_sW] * ne;
        }
        if (within_bounds_2d(iy_sw, ix_sw, inp_H, inp_W)) {
          *out_ptr_NCHW += inp_ptr_NC[iy_sw * inp_sH + ix_sw * inp_sW] * sw;
        }
        if (within_bounds_2d(iy_se, ix_se, inp_H, inp_W)) {
          *out_ptr_NCHW += inp_ptr_NC[iy_se * inp_sH + ix_se * inp_sW] * se;
        }
      }
    } else if (interpolation_mode == torch::detail::GridSamplerInterpolation::Nearest) {
      int ix_nearest = static_cast<int>(roundf(ix));
      int iy_nearest = static_cast<int>(roundf(iy));

      // assign nearest neighor pixel value to output pixel
      auto inp_ptr_NC = inputPtr + n * inp_sN;
      auto out_ptr_NCHW = outputPtr + n * out_sN + h * out_sH + w * out_sW;
      for (int c = 0; c < C; ++c, inp_ptr_NC += inp_sC, out_ptr_NCHW += out_sC) {
        if (within_bounds_2d(iy_nearest, ix_nearest, inp_H, inp_W)) {
          *out_ptr_NCHW = inp_ptr_NC[iy_nearest * inp_sH + ix_nearest * inp_sW];
        } else {
          *out_ptr_NCHW = static_cast<scalar_t>(0);
        }
      }

    }
  }
}

// No shape checking needed here. See # NOTE [ grid_sampler Native Functions ].
int grid_sampler_2d_cuda(int batchSize, const void* inputPtr, const void* gridPtr,
  void* const outputPtr,
  int C,
  int inp_H,
  int inp_W,
  int out_H, // same as grid_H
  int out_W, // same as grid_W
  int inp_sN,
  int inp_sC,
  int inp_sH,
  int inp_sW,
  int grid_sN,
  int grid_sH,
  int grid_sW,
  int grid_sCoor,
  int out_sN,
  int out_sC,
  int out_sH,
  int out_sW,
  torch::detail::GridSamplerInterpolation interpolation_mode,
  torch::detail::GridSamplerPadding padding_mode,
  bool align_corners, torch::detail::GridSamplerDataType dataType, hipStream_t stream)
{

  int count = out_H * out_W * batchSize; // will have inner loop over C
  if (count > 0) {
    if (dataType == torch::detail::GridSamplerDataType::GHALF)
    {
      grid_sampler_2d_kernel<half>
        <<<GET_BLOCKS(count), CUDA_NUM_THREADS, 0, stream>>>(
          count,
          static_cast<const half*>(inputPtr), static_cast<const half*>(gridPtr),
          static_cast<half*>(outputPtr),
          C, inp_H, inp_W, out_H, out_W, inp_sN, inp_sC, inp_sH, inp_sW,
          grid_sN, grid_sH, grid_sW, grid_sCoor, out_sN, out_sC, out_sH, out_sW,
          interpolation_mode,
          padding_mode,
          align_corners);
    }
    else
    {
      grid_sampler_2d_kernel<float>
        <<<GET_BLOCKS(count), CUDA_NUM_THREADS, 0, stream>>>(
          count,
          static_cast<const float*>(inputPtr), static_cast<const float*>(gridPtr),
          static_cast<float*>(outputPtr),
          C, inp_H, inp_W, out_H, out_W, inp_sN, inp_sC, inp_sH, inp_sW,
          grid_sN, grid_sH, grid_sW, grid_sCoor, out_sN, out_sC, out_sH, out_sW,
          interpolation_mode,
          padding_mode,
          align_corners);
    }
  }

  return hipGetLastError() != hipSuccess;
}

